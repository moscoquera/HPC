#include "stdlib.h"
#include "stdio.h"
#include <math.h>
#include <hip/hip_runtime.h>

const int max_val=100;

void generateArray(float* data, int size);

__global__
void vectAddKernel(float* A, float* B, float* C, int n){
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  if (i<n){
    *(C+i)=*(A+i)+*(B+i);
  }
}

void vectorAdd(float* A, float* B,float* C, int n){
  int size=sizeof(float)*n;
  

  float* d_A;
  float* d_B;
  float* d_C;
  
  int _sa = hipMalloc((void**)(&d_A),size);
  int _sb = hipMalloc((void**)(&d_B),size);
  int _sc = hipMalloc((void**)(&d_C),size);

  int _cma=hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  int _cmb=hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  
  dim3 grid (ceil(n/256.0), 1, 1); 
  dim3 block (256, 1, 1);

  vectAddKernel<<<grid,block>>>(d_A,d_B,d_C,n);
  int _cmc=hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
  

  hipFree((void**)&d_A);
  hipFree((void**)&d_B);
  hipFree((void**)&d_C);

}


int main(int argc, char* argv[]){
    if (argc != 2){
      printf("Numero incorrecto de argumentos\n");
      return -1;
    }
    int n = atoi(argv[1]);
    
    float* arr1 = (float*)malloc(sizeof(float)*n);
    float* arr2 = (float*)malloc(sizeof(float)*n);
    float* res = (float*)malloc(sizeof(float)*n);
    generateArray(arr1,n);
    generateArray(arr2,n);

    vectorAdd(arr1,arr2,res,n);
    /*
    printf("Array 1:");
    for(int i=0;i<n;i++){
      printf(" %f",*(arr1+i));
    }
    printf("\n");
    
    printf("Array 2:");
    for(int i=0;i<n;i++){
      printf(" %f",*(arr2+i));
    }
    printf("\n");
    
    
    printf("Res:");
    for(int i=0;i<n;i++){
      printf(" %f",*(res+i));
    }
    printf("\n");*/

}

void generateArray(float* data, int size){
  for(int i=0;i<size;i++){
    *(data+i)=rand() % max_val;
  }
}

