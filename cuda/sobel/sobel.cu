#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <cmath>
#include <hip/hip_runtime.h>


using namespace std;
using namespace cv;



int* _filter(int* data,int channels, int rows,int cols,float *kernel,int kerneldim, int kernelNormalize, int outputNormalizationMode);
uchar * filter(uchar * data,int channels, int rows,int cols,float *kernel,int kerneldim, int kernelNormalize, int outputNormalizationMode);
__device__ int getGlobalIdx_3D_3D();
__device__ int getblockthreadIdx();


int* uchartoint(uchar* data, int size){
	int* buff = (int*)malloc(sizeof(int)*size);
	for(int i=0;i<size;i++){
		*(buff+i)=(int)*(data+i);
	}
	return buff;
}


uchar* inttouchar(int* data, int size){
	uchar* buff = (uchar*)malloc(sizeof(uchar)*size);
	for(int i=0;i<size;i++){
		*(buff+i)=(unsigned char)*(data+i);
	}
	return buff;
}


__global__
void convolution(int* data,int* buff,float* kernel,int* outputvars,int rows,int cols,int channels,int kerneldim){
	int idx = getGlobalIdx_3D_3D();
	int kernelmid;
	extern __shared__ float sharedKernel[];
	float *kernelCenter;

	if (getblockthreadIdx()<kerneldim*kerneldim){
		*(sharedKernel+getblockthreadIdx())=*(kernel+getblockthreadIdx());
	}

	__syncthreads();
/*	
	if (getblockthreadIdx()<kerneldim*kerneldim){
		printf("%d %f\n",getblockthreadIdx(),*(sharedKernel+getblockthreadIdx()));
	}

	__syncthreads();
*/
	kernelmid = kerneldim%2==1?kerneldim/2:(kerneldim-1)/2;
	kernelCenter=sharedKernel+(((kerneldim+1)*kernelmid));
	int row = idx / (cols*channels);
	int col = (idx%(cols*channels))/channels;
	float value=0;
	int pixel=0;
	float kernelVal=0;
	int pixelmin=INT_MAX,pixelmax=INT_MIN;
	int kernelmidHalf=(kerneldim/2);
	if (col>0 && row>0 && row<rows-1 && col<cols-1){
		data = data+idx;
		//r<=(kernelmidHalf) no funciona, no sé porque, pero cuda y yo tenemos un problema.
		for(int r = (-1*kernelmidHalf); r<(kernelmidHalf+1);r++){
			for(int c = -1*kernelmidHalf; c<(kernelmidHalf+1);c++){
				pixel=*(data+(r*cols*channels)+(c*channels));
				kernelVal=*(kernelCenter+(r*-1*kerneldim)+(c*-1));
				value+=kernelVal*pixel;
				if (pixel<pixelmin){
					pixelmin=pixel;			
				}
				if (pixel>pixelmax){
					pixelmax=pixel;			
				}
				
				
			}

		}
		*(buff+idx)=value;

		atomicMin(outputvars,value);	
		atomicMax(outputvars+1,value);
		atomicMin(outputvars+2,pixelmin);	
		atomicMax(outputvars+3,pixelmax);
	
	}	
	//__syncthreads();

	/*if (col>0 && row>0 && row<rows-1 && col<cols-1 && getblockthreadIdx()==0){
		printf("%d %d %d %d\n",*(outputvars),*(outputvars+1),*(outputvars+2),*(outputvars+3));
	}*/

}

__global__
void normalize(int* data,int channels, int rows, int cols,int min, int max, int newMin, int newMax, int mode){
	int pixval=0;
	int i = getGlobalIdx_3D_3D();
	int row = i / (cols*channels);
	int col = (i%(cols*channels))/channels;
	if (row>0 && col>0 && row<rows-1 && col<cols-1){
		pixval=*(data+i);
		if (mode==1){
			*(data+i)=(pixval-min)*((newMax-newMin*1.0)/(max-min))+newMin;
		}else{
			*(data+i)=pixval>newMax?newMax:pixval<newMin?newMin:pixval;
		}
	}
//	__syncthreads();

}


uchar * edge1(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{1,0,-1},
		{0,0,0},
		{-1,0,1}
	};

	return filter(data,channels,rows,cols,(float*)kernel,3,0,0);
}


uchar * edge2(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{0,1,0},
		{1,-4,1},
		{0,1,0}
	};
	
 	return filter(data,channels,rows,cols,(float*)kernel,3,0,0);

}


uchar * edge3(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{-1,-1,-1},
		{-1,8,-1},
		{-1,-1,-1},
	};
	
	return filter(data,channels,rows,cols,(float*)kernel,3,0,0);
}

uchar * sharpen(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{0,-1,0},
		{-1,5,-1},
		{0,-1,0},
	};

	return filter(data,channels,rows,cols,(float*)kernel,3,0,0);

}

uchar * boxblur(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{1,1,1},
		{1,1,1},
		{1,1,1},
	};

	return filter(data,channels,rows,cols,(float*)kernel,3,1,0);
	
}

uchar * gaussianblur(uchar* data,int channels, int rows,int cols){
	float kernel[3][3]={
		{1,2,1},
		{2,4,2},
		{1,2,1},
	};

	return filter(data,channels,rows,cols,(float*)kernel,3,1,0);

}

int * _sobelx(int* data,int channels, int rows,int cols, int mode){
	float kernel[3][3]={
		{1,0,-1},
		{2,0,-2},
		{1,0,-1},
	};
	float * d_kernel;
	hipMalloc(&d_kernel,sizeof(float)*3*3);
	hipMemcpy(d_kernel,(float*)kernel,sizeof(float)*3*3,hipMemcpyHostToDevice);
	int* res = _filter(data,channels,rows,cols,d_kernel,3,0,mode);
	hipFree(d_kernel);
	return res;
	
	
	
}
uchar * sobelx(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data;
	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);
	int* d_output = _sobelx(d_data,channels,rows,cols,0);
	int* output = (int*)malloc(sizeof(int)*rows*cols*channels);
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(d_data);
	hipFree(d_output);
	free(output);
	return out;
}

int * _sobely(int* data,int channels, int rows,int cols, int mode){
	float kernel[3][3]={
		{1,2,1},
		{0,0,0},
		{-1,-2,-1},
	};
	float * d_kernel;
	hipMalloc(&d_kernel,sizeof(float)*3*3);
	hipMemcpy(d_kernel,(float*)kernel,sizeof(float)*3*3,hipMemcpyHostToDevice);
	int* res = _filter(data,channels,rows,cols,d_kernel,3,0,mode);
	hipFree(d_kernel);
	return res;
	
}

int * _sobelx10(int* data,int channels, int rows,int cols, int mode){
	float kernel[3][3]={
		{3,0,-3},
		{10,0,-10},
		{3,0,-3},
	};

	float * d_kernel;
	hipMalloc(&d_kernel,sizeof(float)*3*3);
	hipMemcpy(d_kernel,(float*)kernel,sizeof(float)*3*3,hipMemcpyHostToDevice);
	int* res = _filter(data,channels,rows,cols,d_kernel,3,0,mode);
	hipFree(d_kernel);
	return res;
	
}

int * _sobely10(int* data,int channels, int rows,int cols, int mode){
	float kernel[3][3]={
		{3,10,3},
		{0,0,0},
		{-3,-10,-3},
	};

	float * d_kernel;
	hipMalloc(&d_kernel,sizeof(float)*3*3);
	hipMemcpy(d_kernel,(float*)kernel,sizeof(float)*3*3,hipMemcpyHostToDevice);
	int* res = _filter(data,channels,rows,cols,d_kernel,3,0,mode);
	hipFree(d_kernel);
	return res;
	
}


uchar * sobely(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data;
	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);
	int* d_output = _sobely(d_data,channels,rows,cols,0);
	int* output = (int*)malloc(sizeof(int)*rows*cols*channels);
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(d_data);
	hipFree(d_output);
	free(output);
	return out;
}

__global__
void sobelKernel(int *a, int*b,int* output,int* outputvars,int n){
	int i = getGlobalIdx_3D_3D();
	
	if (i>=n){return;}
	int val=sqrtf((*(a+i))*(*(a+i))+(*(b+i))*(*(b+i)));
	*(output+i)=val;
	atomicMin(outputvars,val);	
	atomicMax(outputvars+1,val);
}

uchar * sobel(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data,*minmaxs;
	int * d_output,*output;

	hipMalloc(&minmaxs,sizeof(int)*2);

	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);


	hipMalloc(&d_output,sizeof(int)*rows*cols*channels);
	output = (int*)malloc(sizeof(int)*rows*cols*channels);
	

	int * filterx =  _sobelx(d_data,channels,rows,cols,-1);
	int * filtery =  _sobely(d_data,channels,rows,cols,-1);
	hipMemset(minmaxs,INT_MAX,1);
	hipMemset(minmaxs+1,INT_MIN,1);
	sobelKernel<<<ceil((rows*cols*channels)/256.0),256>>>(filterx,filtery,d_output,minmaxs,rows*cols*channels);
	int* tmpMinMax = (int*)malloc(sizeof(int)*2);	
	hipMemcpy(tmpMinMax,minmaxs,sizeof(int)*2, hipMemcpyDeviceToHost);

	normalize<<<ceil((rows*cols*channels)/256.0),256>>>(d_output,channels,rows,cols,*(tmpMinMax),*(tmpMinMax+1),0,255,1);	
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	//printf("%d %d %d %d\n",*(tmpMinMax),*(tmpMinMax+1),0,255);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(minmaxs);
	hipFree(d_data);
	hipFree(d_output);
	free(datai);
	free(output);
	hipFree(filterx);
	hipFree(filtery);
	free(tmpMinMax);
	return out;


}


uchar * sobel10(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data,*minmaxs;
	int * d_output,*output;

	hipMalloc(&minmaxs,sizeof(int)*2);

	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);


	hipMalloc(&d_output,sizeof(int)*rows*cols*channels);
	output = (int*)malloc(sizeof(int)*rows*cols*channels);
	

	int * filterx =  _sobelx10(d_data,channels,rows,cols,-1);
	int * filtery =  _sobely10(d_data,channels,rows,cols,-1);
	hipMemset(minmaxs,INT_MAX,1);
	hipMemset(minmaxs+1,INT_MIN,1);
	sobelKernel<<<ceil((rows*cols*channels)/256.0),256>>>(filterx,filtery,d_output,minmaxs,rows*cols*channels);
	int* tmpMinMax = (int*)malloc(sizeof(int)*2);	
	hipMemcpy(tmpMinMax,minmaxs,sizeof(int)*2, hipMemcpyDeviceToHost);

	normalize<<<ceil((rows*cols*channels)/256.0),256>>>(d_output,channels,rows,cols,*(tmpMinMax),*(tmpMinMax+1),0,255,1);	
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	//printf("%d %d %d %d\n",*(tmpMinMax),*(tmpMinMax+1),0,255);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(minmaxs);
	hipFree(d_data);
	hipFree(d_output);
	free(datai);
	free(output);
	hipFree(filterx);
	hipFree(filtery);
	free(tmpMinMax);
	return out;
	 
}

uchar * sobely10(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data;
	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);
	int* d_output = _sobely10(d_data,channels,rows,cols,0);
	int* output = (int*)malloc(sizeof(int)*rows*cols*channels);
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(d_data);
	hipFree(d_output);
	free(output);
	return out;
}

uchar * sobelx10(uchar* data,int channels, int rows,int cols){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data;
	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);
	int* d_output = _sobelx10(d_data,channels,rows,cols,0);
	int* output = (int*)malloc(sizeof(int)*rows*cols*channels);
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	uchar* out = inttouchar(output,rows*cols*channels);
	hipFree(d_data);
	hipFree(d_output);
	free(output);
	return out;
	
}


__global__
void kernelNormAdd(float* kernel,float* output, int kernelNormalize){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float kernelVal=*((float*)kernel+i);

	atomicAdd(output+(kernelVal>=0),kernelVal);
	__syncthreads();
	if (kernelNormalize==1){
		*(kernel+i)=kernelVal/(*output+*(output+1));
	}else{
		*(kernel+i)=kernelVal/(*(output+(kernelVal>=0)));
	}
	__syncthreads();
	
}


int* _filter(int* data,int channels, int rows,int cols,float *kernel,int kerneldim, int kernelNormalize, int outputNormalizationMode){
	int* buff,*minmaxs;
	hipMalloc(&buff,sizeof(int)*channels*rows*cols);
	hipMalloc(&minmaxs,sizeof(int)*4);
	hipMemset(buff,0,sizeof(int)*channels*rows*cols);
	if (kernelNormalize){
		float* sumKernel;
		hipMalloc(&sumKernel,sizeof(float)*2);
		hipMemset(sumKernel,0,sizeof(float)*2);
		kernelNormAdd<<<1,9>>>(kernel,sumKernel,kernelNormalize);
		hipFree(sumKernel);
	}
	int N = rows*cols*channels;
	int ssize = (sizeof(float)*kerneldim*kerneldim);
	hipMemset(minmaxs,INT_MAX,1);
	hipMemset(minmaxs+1,INT_MIN,1);
	hipMemset(minmaxs+2,INT_MAX,1);
	hipMemset(minmaxs+3,INT_MIN,1);
	printf("%f\n",ceil(N/512.0));
	convolution<<<ceil(N/512.0),512,ssize>>>(data,buff,kernel,minmaxs,rows,cols,channels,kerneldim);
	hipError_t err=hipGetLastError();
	if ( hipSuccess !=  err ){
	    printf( "Error!\n" );
            printf("GPUassert: %s\n", hipGetErrorString(err));
	}
	if (outputNormalizationMode>=0){
		  int* tmpMinMax = (int*)malloc(sizeof(int)*4);	
		  hipMemcpy(tmpMinMax,minmaxs,sizeof(int)*4, hipMemcpyDeviceToHost);
		  //printf("%d %d %d %d\n",*(tmpMinMax),*(tmpMinMax+1),*(tmpMinMax+2),*(tmpMinMax+3));
		  normalize<<<ceil(N/256),256>>>(buff,channels,rows,cols,*(tmpMinMax),*(tmpMinMax+1),*(tmpMinMax+2),*(tmpMinMax+3),outputNormalizationMode);
		free(tmpMinMax);
	}
	hipFree(minmaxs);
	return buff;
}

uchar* filter(uchar* data,int channels, int rows,int cols,float *kernel,int kerneldim, int kernelNormalize, int outputNormalizationMode){
	int* datai = uchartoint(data,channels*rows*cols);
	int * d_data;
	hipMalloc(&d_data,sizeof(int)*channels*rows*cols);
	hipMemcpy(d_data,datai,sizeof(int)*channels*rows*cols,hipMemcpyHostToDevice);
	float * d_kernel;
	hipMalloc(&d_kernel,sizeof(float)*3*3);
	hipMemcpy(d_kernel,kernel,sizeof(float)*3*3,hipMemcpyHostToDevice);
	int* d_output = _filter(d_data,channels,rows,cols,d_kernel,kerneldim,kernelNormalize,outputNormalizationMode);
	int* output = (int*)malloc(sizeof(int)*rows*cols*channels);
	hipMemcpy(output,d_output,sizeof(int)*rows*cols*channels, hipMemcpyDeviceToHost);
	uchar* out = inttouchar(output,rows*cols*channels);

	hipFree(d_data);
	hipFree(d_output);
	free(datai);
	free(output);
	return out;
}


int main(int argc, char** argv){


	if (argc<3){
		cout<<"./nombre imagen filtro"<<endl;
		return 0;
	}


	char* nfiltro=*(argv+2);
	uchar* (*filtro)(uchar*,int,int,int)=0;

	if(strcmp(nfiltro,"sobel")==0) filtro=sobel;
	if(strcmp(nfiltro,"sobelx")==0) filtro=sobelx;
	if(strcmp(nfiltro,"sobely")==0) filtro=sobely;
	if(strcmp(nfiltro,"sobel10")==0) filtro=sobel10;
	if(strcmp(nfiltro,"sobelx10")==0) filtro=sobelx10;
	if(strcmp(nfiltro,"sobely10")==0) filtro=sobely10;
	if(strcmp(nfiltro,"edge1")==0) filtro=edge1;
	if(strcmp(nfiltro,"edge2")==0) filtro=edge2;
	if(strcmp(nfiltro,"edge3")==0) filtro=edge3;
	if(strcmp(nfiltro,"boxblur")==0) filtro=boxblur;
	if(strcmp(nfiltro,"gaussianblur")==0) filtro=gaussianblur;
	if(strcmp(nfiltro,"sharpen")==0) filtro=sharpen;
	if (filtro==0){
		cout<<"metodo erroneo"<<endl;
		return 1;
	}

	Mat image;
	image = imread(*(argv+1), CV_LOAD_IMAGE_COLOR);
	Mat m1;

    
	if(! image.data )                              // Check for invalid input
	{
		cout <<  "Could not open or find the image" << std::endl ;
		return -1;
	}


	m1 = Mat (image);
	m1.data=filtro(image.data,3,image.rows,image.cols);
	namedWindow( "original", WINDOW_AUTOSIZE );
	imshow( "original", image );             
	namedWindow( "filter", WINDOW_AUTOSIZE );
	imshow( "filter", m1 );             

    waitKey();                                        // Wait for a keystroke in the window
  return 0;
}


__device__ int getGlobalIdx_3D_3D()
{
	int blockId = blockIdx.x 
			 + blockIdx.y * gridDim.x 
			 + gridDim.x * gridDim.y * blockIdx.z; 
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
			  + (threadIdx.z * (blockDim.x * blockDim.y))
			  + (threadIdx.y * blockDim.x)
			  + threadIdx.x;
	return threadId;
}


__device__ int getblockthreadIdx(){
	return (threadIdx.z * (blockDim.x * blockDim.y))
			  + (threadIdx.y * blockDim.x)
			  + threadIdx.x;
}



